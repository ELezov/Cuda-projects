#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 15000

using namespace std;
__global__ void MatrVectMul(int *d_c, int *d_a, int *d_b)
{
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i<N)
	{
    		d_c[i]=0;
    		for (int k=0;k<N;k++)
    			d_c[i]+=d_a[i+k*N]*d_b[k];
	}
}
//�����: threadIdx.x � ������������� ������ � ����� �� ���������� x,
//blockIdx.x � ������������� ����� � ����� �� ���������� x,
//blockDim.x � ���������� ������� � ����� �����.

int main()
{
	hipEvent_t start, stop;
	float gpuTime=0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// ������� ������� � ����������� ������
	int *h_a,*h_b,*h_c;
	h_a = new int[N*N];
	h_b = new int[N];
	h_c = new int[N];

	for (int i=0;i<N;i++)	 // ������������� �������� a � b
	{ 
		for (int k=0;k<N;k++)
    		{
          		h_a[i*N+k]=1;
    		}
	  	h_b[i]=2;
	}

	// ��������� �� ������� � �����������
	int *d_a,*d_b,*d_c;

	// ��������� �����������
	hipMalloc((void **)&d_a, sizeof(int)*N*N); 
	hipMalloc((void **)&d_b, sizeof(int)*N); 
	hipMalloc((void **)&d_c, sizeof(int)*N); 

	// ����������� �� ����������� ������ � �����������
	hipMemcpy(d_a, h_a, sizeof(int)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(int)*N, hipMemcpyHostToDevice);

	// ��������� ���������� ������
	dim3 grid((N+255)/256, 1, 1);
	// ��������� ���������� ������� � �����
	dim3 threads(256, 1, 1);
	//������ ������� �������
	hipEventRecord(start,0);
	hipEventSynchronize(start);
	// ����� �������
	MatrVectMul <<< grid, threads >>> (d_c, d_a, d_b);

	 //��������� ������ ����, ��������� �������
	hipEventRecord(stop,0);
	//�������������� � �������� ��������� ��������
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime,start,stop);
	printf("Time: %.9f msec.\n",gpuTime);

	// ����������� �� ����������� � ����������� ������ 
	hipMemcpy(h_c, d_c, sizeof(int)*N, hipMemcpyDeviceToHost);
	
	//for (int i=0;i<N;i++) cout<<h_c[i]<<' ';
	// ������������ ������
	hipFree(d_a); 
	hipFree(d_b); 
	hipFree(d_c);
}
